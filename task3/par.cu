#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void array_ops(float* a, float* b, float* add, float* sub, float* mul, float* divv, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        add[idx] = a[idx] + b[idx];
        sub[idx] = a[idx] - b[idx];
        mul[idx] = a[idx] * b[idx];
        divv[idx] = b[idx] != 0 ? a[idx] / b[idx] : 0;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        printf("Использование: %s <размер массива> <кол-во потоков>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);
    int threads = atoi(argv[2]);
    int blocks = (size + threads - 1) / threads;

    float* h_a = (float*)malloc(size * sizeof(float));
    float* h_b = (float*)malloc(size * sizeof(float));
    for (int i = 0; i < size; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    float *d_a, *d_b, *d_add, *d_sub, *d_mul, *d_div;
    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_add, size * sizeof(float));
    hipMalloc(&d_sub, size * sizeof(float));
    hipMalloc(&d_mul, size * sizeof(float));
    hipMalloc(&d_div, size * sizeof(float));

    hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);

    clock_t start = clock();
    array_ops<<<blocks, threads>>>(d_a, d_b, d_add, d_sub, d_mul, d_div, size);
    hipDeviceSynchronize();
    clock_t end = clock();

    printf("Операции с массивами завершены.\n");
    printf("Время выполнения: %.4f секунд\n", (double)(end - start) / CLOCKS_PER_SEC);

    hipFree(d_a); hipFree(d_b); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(h_a); free(h_b);
    return 0;
}