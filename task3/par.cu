#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void array_ops(float* a, float* b, float* add, float* sub, float* mul, float* divv, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        add[idx] = a[idx] + b[idx];
        sub[idx] = a[idx] - b[idx];
        mul[idx] = a[idx] * b[idx];
        divv[idx] = b[idx] != 0 ? a[idx] / b[idx] : 0;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        printf("Использование: %s <размер массива> <кол-во потоков>\n", argv[0]);
        return 1;
    }

    int size = atoi(argv[1]);
    int threads = atoi(argv[2]);
    int blocks = (size + threads - 1) / threads;

    float *h_a = (float*)malloc(size * sizeof(float));
    float *h_b = (float*)malloc(size * sizeof(float));
    float *d_a, *d_b, *d_add, *d_sub, *d_mul, *d_div;

    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_add, size * sizeof(float));
    hipMalloc(&d_sub, size * sizeof(float));
    hipMalloc(&d_mul, size * sizeof(float));
    hipMalloc(&d_div, size * sizeof(float));

    float totalTime = 0.0f;
    srand(time(NULL));

    for (int iter = 0; iter < 100; iter++) {
        // Генерация случайных данных
        for (int i = 0; i < size; i++) {
            h_a[i] = rand() / (float)RAND_MAX;
            h_b[i] = rand() / (float)RAND_MAX;
        }

        hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);

        // Таймеры CUDA
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        array_ops<<<blocks, threads>>>(d_a, d_b, d_add, d_sub, d_mul, d_div, size);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed;
        hipEventElapsedTime(&elapsed, start, stop);
        totalTime += elapsed;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("Среднее время выполнения на GPU за 100 запусков: %.4f мс\n", totalTime / 100.0f);

    hipFree(d_a); hipFree(d_b); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(h_a); free(h_b);

    return 0;
}
