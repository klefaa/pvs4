#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

// Ядро CUDA: операции с матрицами
__global__ void matrix_ops(float* a, float* b, float* add, float* sub, float* mul, float* divv, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N) {
        int idx = i * N + j;
        add[idx] = a[idx] + b[idx];
        sub[idx] = a[idx] - b[idx];
        mul[idx] = a[idx] * b[idx];
        divv[idx] = b[idx] != 0 ? a[idx] / b[idx] : 0;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        printf("Использование: %s <размерность матрицы NxN> <количество потоков на блок>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);
    int size = N * N;

    // Выбор "квадратной" формы блока
    int blockDimX = (int)sqrt((double)threadsPerBlock);
    while (threadsPerBlock % blockDimX != 0) blockDimX--;
    int blockDimY = threadsPerBlock / blockDimX;

    dim3 threads(blockDimX, blockDimY);
    dim3 blocks((N + blockDimX - 1) / blockDimX, (N + blockDimY - 1) / blockDimY);

    printf("Конфигурация CUDA:\n");
    printf("  blockDim = (%d, %d)\n", blockDimX, blockDimY);
    printf("  gridDim  = (%d, %d)\n", blocks.x, blocks.y);

    float *a = (float*)malloc(size * sizeof(float));
    float *b = (float*)malloc(size * sizeof(float));
    float *d_a, *d_b, *d_add, *d_sub, *d_mul, *d_div;

    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_add, size * sizeof(float));
    hipMalloc(&d_sub, size * sizeof(float));
    hipMalloc(&d_mul, size * sizeof(float));
    hipMalloc(&d_div, size * sizeof(float));

    srand(time(NULL));
    float totalTime = 0.0f;

    for (int run = 0; run < 100; run++) {
        for (int i = 0; i < size; i++) {
            a[i] = rand() / (float)RAND_MAX;
            b[i] = rand() / (float)RAND_MAX;
        }

        hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        matrix_ops<<<blocks, threads>>>(d_a, d_b, d_add, d_sub, d_mul, d_div, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float ms;
        hipEventElapsedTime(&ms, start, stop);
        totalTime += ms;

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("Среднее время выполнения за 100 запусков: %.4f мс\n", totalTime / 100.0f);

    hipFree(d_a); hipFree(d_b); hipFree(d_add); hipFree(d_sub); hipFree(d_mul); hipFree(d_div);
    free(a); free(b);
    return 0;
}
